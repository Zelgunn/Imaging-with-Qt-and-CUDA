#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void raiseImageKernel(float *output, float *input, int width, int height, float factor)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;

	if (i >= width || j >= height) return;

	int idx = j*width + i;

	output[idx] = factor * input[idx];
}

float DllExport *raiseImage(float *h_input, int width, int height, float factor);

__global__ void pixelSubtractionKernel(float *output, float *input1, float *input2, int width, int height)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;

	int idx = j*width + i;

	if (i < width && j < height)
		output[idx] = input1[idx] - input2[idx];
}

float DllExport *pixelSubtraction(float *h_input1, float *h_input2, int width, int height)
{
	int elementCount = width*height;
	int size = elementCount*sizeof(float);

	float *h_output1 = (float*)malloc(size), *h_output2;
	float *dev_input1, *dev_input2, *dev_output;

	hipMalloc((void**)&dev_input1, size);
	hipMalloc((void**)&dev_input2, size);
	hipMalloc((void**)&dev_output, size);

	hipMemcpy(dev_input1, h_input1, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_input2, h_input2, size, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(THREADS_PER_BLOCKDIM, THREADS_PER_BLOCKDIM);
	dim3 blockCount(((width - 1) / THREADS_PER_BLOCKDIM) + 1, ((height - 1) / THREADS_PER_BLOCKDIM) + 1);

	pixelSubtractionKernel << <blockCount, threadsPerBlock >> >(dev_output, dev_input1, dev_input2, width, height);

	hipMemcpy(h_output1, dev_output, size, hipMemcpyDeviceToHost);
	float factor = 0.0;
	for (int i = 0; i < elementCount; i++)
		if (factor < h_output1[i])
			factor = h_output1[i];
	if (factor == 0.0) return h_output1;
	factor = 1.0 / factor;

	raiseImageKernel << <blockCount, threadsPerBlock >> >(dev_input1, dev_output, width, height, factor);
	h_output2 = (float*)malloc(size);
	hipMemcpy(h_output2, dev_input1, size, hipMemcpyDeviceToHost);

	free(h_output1);
	hipFree(dev_input1);
	hipFree(dev_input2);
	hipFree(dev_output);

	return h_output2;
}

__global__ void gaussianBlurKernel(float *output, float *input, int width, int height)
{
	const int radius = 2;

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;

	if (i >= width || j >= height) return;

	int idx = j*width + i;
	int ix, jy;
	float total = 0.0, totalFactor = 0.0, tmpFactor;
	int tmp;

	for (int x = -radius; x < radius; x++)
	{
		ix = i + x;
		if ((ix >= 0) && (ix < width))
		{
			for (int y = -radius; y < radius; y++)
			{
				jy = j + y;
				if ((jy >= 0) && (jy < height))
				{
					tmp = x*x + y*y;
					switch (tmp)
					{
					case 4:			// Apparait 8 fois
						tmpFactor = 5.0;
						break;
					case 5:
						tmpFactor = 4.0;
						break;
					case 8:
						tmpFactor = 2.0;
						break;
					case 1:
						tmpFactor = 12.0;
						break;
					case 2:
						tmpFactor = 9.0;
						break;
					default:		// Apparait 1 fois
						tmpFactor = 15.0;
						break;
					}
					totalFactor += tmpFactor;
					total += tmpFactor*input[ix + jy*width];
				}
			}
		}
	}

	output[idx] = total / totalFactor;
}

float DllExport *gaussianBlur(float *h_input, int width, int height)
{
	int elementCount = height*width;
	int size = elementCount*sizeof(float);

	float *h_output = (float*)malloc(size);
	float *dev_input, *dev_output;

	hipMalloc((void**)&dev_input, size);
	hipMalloc((void**)&dev_output, size);

	hipMemcpy(dev_input, h_input, size, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(THREADS_PER_BLOCKDIM, THREADS_PER_BLOCKDIM);
	dim3 blockCount(((width - 1) / THREADS_PER_BLOCKDIM) + 1, ((height - 1) / THREADS_PER_BLOCKDIM) + 1);

	gaussianBlurKernel << < blockCount, threadsPerBlock >> > (dev_output, dev_input, width, height);

	hipMemcpy(h_output, dev_output, size, hipMemcpyDeviceToHost);

	hipFree(dev_input);
	hipFree(dev_output);

	return h_output;
}

float DllExport *differenceOfGaussian(float *h_input, int width, int height)
{
	return pixelSubtraction(gaussianBlur(h_input, width, height), h_input, width, height);
}

__global__ void underSamplingKernel(float *output, float *input, int newWidth, int newHeight, float sizeFactor)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int width = newWidth*sizeFactor;

	if (i >= newWidth || j >= newHeight) return;

	int idxOutput = j*newWidth + i;
	int idxInput = (j*width + i)*sizeFactor;

	output[idxOutput] = (input[idxInput] + input[idxInput + 1] + input[idxInput + width] + input[idxInput + width +1]) / 4;
}

float DllExport *underSampling(float *h_input, int width, int height, float sizeFactor)
{
	if (sizeFactor < 1.0) return NULL;
	int newWidth = width / sizeFactor;
	int newHeight = height / sizeFactor;

	int elementCount = height*width;
	int newElementCount = newWidth*newHeight;

	int size = elementCount*sizeof(float);
	int newSize = newElementCount*sizeof(float);

	float *h_output = (float*)malloc(newSize);
	float *dev_input, *dev_output;

	hipMalloc((void**)&dev_input, size);
	hipMalloc((void**)&dev_output, newSize);

	hipMemcpy(dev_input, h_input, size, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(THREADS_PER_BLOCKDIM, THREADS_PER_BLOCKDIM);
	dim3 blockCount(((newWidth - 1) / THREADS_PER_BLOCKDIM) + 1, ((newHeight - 1) / THREADS_PER_BLOCKDIM) + 1);

	underSamplingKernel << < blockCount, threadsPerBlock >> > (dev_output, dev_input, newWidth, newHeight, sizeFactor);

	hipMemcpy(h_output, dev_output, newSize, hipMemcpyDeviceToHost);

	hipFree(dev_input);
	hipFree(dev_output);

	return h_output;
}

float DllExport *laplacian4Neib(float *h_input, int width, int height);
float DllExport *applyMask(float *h_input, int width, int height, int radius, float *h_mask);
float DllExport *applyThreshold(float *h_input, int width, int height, int threshold);

